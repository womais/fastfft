
#include <hip/hip_runtime.h>
#include <stdio.h>

struct Complex {
    double real;
    double imag;
};
Complex* a_device = NULL;
Complex* host_mem = NULL;
Complex* device_precomp = NULL;

__global__
void donkey_inv(Complex* precomp,
                Complex* a, 
                int blocks_per_half, 
                int lg_len,
                int num_blocks) {
    const int blk = blockIdx.x * blockDim.x + threadIdx.x;
    if (blk > num_blocks) return;
    const int len = 1 << lg_len;

    // more...
    const int which_half = blk / blocks_per_half;
    const int block_ind = blk % blocks_per_half;
    const int half_start = (which_half << lg_len);
    const int start = half_start + block_ind * 2;

    Complex w[2], u[2], v[2];
    for (int j = 0; j < 2; ++j) {
        w[j] = precomp[len + start - half_start + j];
        u[j] = a[start + j];
        v[j] = a[start + j + (len >> 1)];
        w[j].imag *= -1;
        v[j] = {v[j].real * w[j].real - v[j].imag * w[j].imag,
                v[j].real * w[j].imag + v[j].imag * w[j].real};
        a[start + j].real = u[j].real + v[j].real;
        a[start + j].imag = u[j].imag + v[j].imag;
        a[start + j + (len >> 1)].real = u[j].real - v[j].real;
        a[start + j + (len >> 1)].imag = u[j].imag - v[j].imag;
    }
}

__global__
void donkey(Complex* precomp,
                Complex* a, 
                int blocks_per_half, 
                int lg_len,
                int num_blocks) {
    const int blk = blockIdx.x * blockDim.x + threadIdx.x;
    if (blk > num_blocks) return;
    const int len = 1 << lg_len;

    // more...
    const int which_half = blk / blocks_per_half;
    const int block_ind = blk % blocks_per_half;
    const int half_start = (which_half << lg_len);
    const int start = half_start + block_ind * 2;

    Complex w[2], u[2], v[2];
    for (int j = 0; j < 2; ++j) {
        w[j] = precomp[len + start - half_start + j];
        u[j] = a[start + j];
        v[j] = a[start + j + (len >> 1)];
        v[j] = {v[j].real * w[j].real - v[j].imag * w[j].imag,
                v[j].real * w[j].imag + v[j].imag * w[j].real};
        a[start + j].real = u[j].real + v[j].real;
        a[start + j].imag = u[j].imag + v[j].imag;
        a[start + j + (len >> 1)].real = u[j].real - v[j].real;
        a[start + j + (len >> 1)].imag = u[j].imag - v[j].imag;
    }
}

// costly, but should only do it once...
// this copies over precomputed roots of unity
// to global memory.
extern "C"
{
void initialize_gpu_precomp(size_t N, void* data) {
    hipMalloc(&device_precomp, N * sizeof(Complex));
    hipMemcpy(device_precomp, data, N * sizeof(Complex), hipMemcpyHostToDevice);
}

void initialize_gpu_data(size_t N, void* values) {
    if (a_device == NULL)
        hipMalloc(&a_device, N * sizeof(Complex));
    host_mem = (Complex*)values;
    hipMemcpy(a_device, host_mem, N * sizeof(Complex), hipMemcpyHostToDevice);
}

void finish_gpu_data(size_t N) {
    hipMemcpy(host_mem, a_device, N * sizeof(Complex), hipMemcpyDeviceToHost);
    hipFree(a_device);
    a_device = NULL;
}
void run_gpu_pass(int len, int lg_len, int n) {
    const int num_half_intervals = n >> lg_len;
    const int blocks_per_half = (len >> 2);
    const int num_blocks = blocks_per_half * num_half_intervals; 
    donkey<<<(num_blocks + 255) / 512, 512>>>(device_precomp, a_device, blocks_per_half, lg_len, num_blocks);
    hipDeviceSynchronize();

}
void run_gpu_pass_inv(int len, int lg_len, int n) {
    const int num_half_intervals = n >> lg_len;
    const int blocks_per_half = (len >> 2);
    const int num_blocks = blocks_per_half * num_half_intervals; 
    donkey_inv<<<(num_blocks + 255) / 512, 512>>>(device_precomp, a_device, blocks_per_half, lg_len, num_blocks);
    hipDeviceSynchronize();

}

}
